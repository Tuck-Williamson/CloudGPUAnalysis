#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <stdio.h>

#define cuEChk(toCheck) errChk(toCheck, __LINE__)
void errChk(hipError_t status, size_t line){
    if(status != hipSuccess){
        std::cerr << "There was a cuda error at line " << line << "." << std::endl;
        std::cerr << "Error (" << status << "): " << hipGetErrorName(status) << "::" << hipGetErrorString(status) << std::endl;
        throw 1;
    }
}

__device__ char* devMessage= "\t\tHello from GPU.\n";

//template <typename TK>
#define TK unsigned int
__global__ void k(TK *d, size_t n){

  if(threadIdx.x == 0 && blockIdx.x == 0){
    printf("\tTest message: %s\n", devMessage);
    printf("\tMessage Location: %p\n", devMessage);
  }

  TK *myCode = (TK*)&devMessage[0];
  for (size_t i = blockIdx.x*blockDim.x+threadIdx.x; i < n; i+=gridDim.x*blockDim.x)
    d[i] = myCode[i];
}

template <typename T>
void printMem(T *d, size_t n){
  //std::cout << std::hex;
  for(uint i = 0; i < n; i++){
    //std::cout << "(" << std::dec << i << "): " << std::hex << d[i]  << " ";
    printf("(%6d): %08x ", i, d[i]);
    if( (i+1) % 8 == 0 ){
        //std::cout << std::endl;
        printf("\n");
    }
  }
}

char gc(char curChar){
  if(31 < curChar && curChar < 127)
    return curChar;
  else
    return '.';
}

template <typename T>
void printMemChar(T *d, size_t n, uint numEleNLine){
  //std::cout << std::hex;
  for(uint i = 0; i < n; i+=numEleNLine){
    char* data = (char*)&d[i];
    unsigned int dIdx = 0;
    printf("(%6d): ", i);
    for ( uint curIdx = 0; curIdx < numEleNLine; curIdx++){
        printf("%08x ", d[i+curIdx]);
    }
    printf("|");
    for ( uint curIdx = 0; curIdx < numEleNLine; curIdx++){
        printf("%c%c%c%c", gc(data[dIdx]), gc(data[dIdx+1]), gc(data[dIdx+2]), gc(data[dIdx+3]));
        dIdx += 4;
    }

    printf("\n");
  }
}


int main(){

  unsigned int *d;
  size_t n = 1*1024;
  hipHostAlloc(&d, sizeof(d[0])*n, hipHostMallocDefault);
  //k<<<160, 1024>>>(d, n);
  k<<<1, 1024>>>(d, n);
  hipDeviceSynchronize();
  std::cout << "Print before dev to host memcopy.\n";
  printMemChar(d, n, 8);
  std::cout << "\n\n";

}