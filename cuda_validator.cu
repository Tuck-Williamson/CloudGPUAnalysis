//From https://developer.nvidia.com/blog/even-easier-introduction-cuda/
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>

#define cuEChk(toCheck) errChk(toCheck, __LINE__)
void errChk(hipError_t status, size_t line){
    if(status != hipSuccess){
        std::cerr << "There was a cuda error at line " << line << "." << std::endl;
        std::cerr << "Error (" << status << "): " << hipGetErrorName(status) << "::" << hipGetErrorString(status) << std::endl;
        throw 1;
    }
}

// Kernel function to add the elements of two arrays
__global__ void validation_gen(const unsigned int maxIdx, unsigned int* dataPtr)
{
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int idx = index; idx < maxIdx; idx += stride)
      dataPtr[idx] = stride << 16 | index;
}

int main(int argc, char *argv[])
{
  unsigned int N = 256<<2;
  unsigned int* dataPtr;
  const size_t memSize = N*sizeof(dataPtr[0]);

  // Allocate Unified Memory - accessible from CPU or GPU
//   cudaMallocManaged(&dataPtr, N*sizeof(dataPtr[0]));
  cuEChk(hipMalloc(&dataPtr, memSize));

  // Run kernel on 1M elements on the GPU
  validation_gen<<<1, 256>>>(N, dataPtr);

  // Wait for GPU to finish before accessing on host
  cuEChk(hipDeviceSynchronize());

  unsigned short* hostPtr = (unsigned short*)malloc(memSize);
  cuEChk(hipMemcpy((void*)hostPtr, dataPtr, memSize, hipMemcpyDeviceToHost));

  std::cout << std::hex;
  for(uint i = 0; i < N; i++){
    auto idx = i % 256;
    auto stride = i / 256;

    std::cout << "(" << std::dec << i << "): " << std::hex << hostPtr[i*2] << hostPtr[i*2+1] << " -> " << stride << idx << " ";
    if(i % 4 == 0){
        std::cout << std::endl;
    }
  }

  std::cout << std::endl << std::endl;

  // Free memory
  cuEChk(hipFree(dataPtr));

  return 0;
}